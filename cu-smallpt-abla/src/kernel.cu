#include "hip/hip_runtime.h"
//-----------------------------------------------------------------------------
// Includes
//-----------------------------------------------------------------------------
#pragma region

#include "imageio.hpp"
#include "sampling.cuh"
#include "specular.cuh"
#include "sphere.hpp"

#include "cuda_tools.hpp"
#include <chrono>

#pragma endregion

//-----------------------------------------------------------------------------
// Defines
//-----------------------------------------------------------------------------
#pragma region

#define REFRACTIVE_INDEX_OUT 1.0
#define REFRACTIVE_INDEX_IN  1.5

#pragma endregion

//-----------------------------------------------------------------------------
// Declarations and Definitions
//-----------------------------------------------------------------------------
namespace smallpt {

	//__constant__ Sphere dev_spheres[9];

	const Sphere g_spheres[] = {
		Sphere(1e5,  Vector3(1e5 + 1, 40.8, 81.6),   Vector3(),   Vector3(0.15,0.85,0.25), Reflection_t::Diffuse),	 //Left
		Sphere(1e5,  Vector3(-1e5 + 99, 40.8, 81.6), Vector3(),   Vector3(0.13,0.73,0.81), Reflection_t::Diffuse),	 //Right
		Sphere(1e5,  Vector3(50, 40.8, 1e5),         Vector3(),   Vector3(0.75),           Reflection_t::Diffuse),	 //Back
		Sphere(1e5,  Vector3(50, 40.8, -1e5 + 170),  Vector3(),   Vector3(0.75),               Reflection_t::Diffuse),	 //Front
		Sphere(1e5,  Vector3(50, 1e5, 81.6),         Vector3(),   Vector3(0.75),           Reflection_t::Diffuse),	 //Bottom
		Sphere(1e5,  Vector3(50, -1e5 + 81.6, 81.6), Vector3(),   Vector3(0.75),           Reflection_t::Diffuse),	 //Top
		Sphere(16.5, Vector3(27, 16.5, 47),          Vector3(),   Vector3(0.999),          Reflection_t::Specular),	 //Mirror
		Sphere(16.5, Vector3(73, 16.5, 78),          Vector3(),   Vector3(0.999),          Reflection_t::Diffuse),//Glass
		Sphere(8.5, Vector3(73, 53.5, 78),          Vector3(),   Vector3(0.999),          Reflection_t::Refractive),//Glass
		Sphere(600,	 Vector3(50, 681.6 - .27, 81.6), Vector3(12), Vector3(),               Reflection_t::Diffuse)	 //Light
	};

	__device__ inline bool Intersect(const Sphere* dev_spheres, 
									 std::size_t nb_spheres, 
									 const Ray& ray, 
									 size_t& id) {
		
		bool hit = false;
		for (std::size_t i = 0u; i < nb_spheres; ++i) {
			if (dev_spheres[i].Intersect(ray)) {
				hit = true;
				id  = i;
			}
		}

		return hit;
	}

	__device__ static Vector3 Radiance(const Sphere* dev_spheres, 
									   std::size_t nb_spheres,
									   const Ray& ray, 
									   hiprandState* state) {
		
		Ray r = ray;
		Vector3 L;
		Vector3 F(1.0);

		while (true) {
			std::size_t id;
			if (!Intersect(dev_spheres, nb_spheres, r, id)) {
				return L;
			}

			const Sphere& shape = dev_spheres[id];
			const Vector3 p = r(r.m_tmax);
			const Vector3 n = Normalize(p - shape.m_p);

			L += F * shape.m_e;
			F *= shape.m_f;

			// Russian roulette
			if (4 < r.m_depth) {
				const double continue_probability = shape.m_f.Max();
				if (hiprand_uniform_double(state) >= continue_probability) {
					return L;
				}
				F /= continue_probability;
			}

			// Next path segment
			switch (shape.m_reflection_t) {
			
			case Reflection_t::Specular: {
				const Vector3 d = IdealSpecularReflect(r.m_d, n);
				r = Ray(p, d, EPSILON_SPHERE, INFINITY, r.m_depth + 1u);
				break;
			}
			
			case Reflection_t::Refractive: {
				double pr;
				const Vector3 d = IdealSpecularTransmit(r.m_d, n, REFRACTIVE_INDEX_OUT, REFRACTIVE_INDEX_IN, pr, state);
				F *= pr;
				r = Ray(p, d, EPSILON_SPHERE, INFINITY, r.m_depth + 1u);
				break;
			}
			
			default: {
				const Vector3 w = (0.0 > n.Dot(r.m_d)) ? n : -n;
				const Vector3 u = Normalize((abs(w.m_x) > 0.1 ? Vector3(0.0, 1.0, 0.0) : Vector3(1.0, 0.0, 0.0)).Cross(w));
				const Vector3 v = w.Cross(u);

				const Vector3 sample_d = CosineWeightedSampleOnHemisphere(hiprand_uniform_double(state), hiprand_uniform_double(state));
				const Vector3 d = Normalize(sample_d.m_x * u + sample_d.m_y * v + sample_d.m_z * w);
				r = Ray(p, d, EPSILON_SPHERE, INFINITY, r.m_depth + 1u);
			}
			}
		}
	}

	__global__ static void kernel(const Sphere* dev_spheres, 
								  std::size_t nb_spheres,
								  std::uint32_t w, 
								  std::uint32_t h, 
								  Vector3* Ls, 
								  std::uint32_t nb_samples,
								  std::uint32_t k) {
		
		const std::uint32_t base_x = (threadIdx.x + blockIdx.x * blockDim.x)*k;
		const std::uint32_t base_y = (threadIdx.y + blockIdx.y * blockDim.y)*k;

		const Vector3 eye = { 50.0, 52.0, 295.6 };
		const Vector3 gaze = Normalize(Vector3(0.0, -0.042612, -1.0));
		const double fov = 0.5135;
		const Vector3 cx = { w * fov / h, 0.0, 0.0 };
		const Vector3 cy = Normalize(cx.Cross(gaze)) * fov;

		for (std::uint32_t i = 0; i < k*k; ++i) {
			int x = base_x + i % k;
			int y = base_y + i / k;
			int pixel_index = (h - 1u - y) * w + x;

			if (x >= w || y >= h) {
				continue;
			}
			// hiprand state 初始化
			hiprandState state;
			hiprand_init(pixel_index, 0u, 0u, &state);

			for (std::size_t sy = 0u; sy < 2u; ++sy) {
				for (std::size_t sx = 0u; sx < 2u; ++sx) {
					Vector3 L;
					for (std::size_t s = 0u; s < nb_samples; ++s) {
						double u1 = 2.0 * hiprand_uniform_double(&state);
						double u2 = 2.0 * hiprand_uniform_double(&state);
						double dx = (u1 < 1.0) ? sqrt(u1) - 1.0 : 1.0 - sqrt(2.0 - u1);
						double dy = (u2 < 1.0) ? sqrt(u2) - 1.0 : 1.0 - sqrt(2.0 - u2);
						Vector3 d = cx * (((sx + 0.5 + dx) * 0.5 + x) / w - 0.5) +
									cy * (((sy + 0.5 + dy) * 0.5 + y) / h - 0.5) + gaze;
						L += Radiance(dev_spheres, nb_spheres,
									Ray(eye + d * 130, Normalize(d), EPSILON_SPHERE), &state) * (1.0 / nb_samples);
					}
					Ls[pixel_index] += 0.25 * Clamp(L);
				}
			}
		}
	}

	static void Render(std::uint32_t nb_samples, dim3 nthreads, dim3 nblocks, FILE* log_fp, int pixels_per_thread) {
		const std::uint32_t w = 1024u;
		const std::uint32_t h = 768u;
		const std::uint32_t nb_pixels = w * h;

		// Set up device memory
		//HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(dev_spheres), spheres, sizeof(spheres)) );
		Sphere* dev_spheres;
		HANDLE_ERROR(hipMalloc((void**)&dev_spheres, sizeof(g_spheres)));
		HANDLE_ERROR(hipMemcpy(dev_spheres, g_spheres, sizeof(g_spheres), hipMemcpyHostToDevice));
		Vector3* dev_Ls;
		HANDLE_ERROR(hipMalloc((void**)&dev_Ls, nb_pixels * sizeof(Vector3)));
		HANDLE_ERROR(hipMemset(dev_Ls, 0, nb_pixels * sizeof(Vector3)));

		// ===== 计时开始 =====
		auto total_start = std::chrono::high_resolution_clock::now();

		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);
		kernel<<< nblocks, nthreads >>>(dev_spheres, sizeof(g_spheres) / sizeof(g_spheres[0]), w, h, dev_Ls, nb_samples, sqrt(pixels_per_thread));

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		float kernel_time_ms = 0.0f;
		hipEventElapsedTime(&kernel_time_ms, start, stop);
		hipEventDestroy(start);
		hipEventDestroy(stop);

		auto total_end = std::chrono::high_resolution_clock::now();
		float total_time_ms = std::chrono::duration<float, std::milli>(total_end - total_start).count();

		float efficiency = kernel_time_ms / total_time_ms;
		std::printf("[Block=%ux%u] Pixel_per_thread=%u Samples=%u | Kernel: %.2f ms | Total: %.2f ms | Eff: %.3f\n",
					nthreads.x, nthreads.y, pixels_per_thread, nb_samples, kernel_time_ms, total_time_ms, efficiency);

		// 写入 CSV 日志
		if (log_fp) {
			std::fprintf(log_fp, "%u,%u,%u,%.3f,%.3f,%.3f\n", 
             nthreads.x, pixels_per_thread, nb_samples, kernel_time_ms, total_time_ms, efficiency);
			std::fflush(log_fp);
		}

		// Set up host memory
		Vector3* Ls = (Vector3*)malloc(nb_pixels * sizeof(Vector3));
		// Transfer device -> host
		HANDLE_ERROR(hipMemcpy(Ls, dev_Ls, nb_pixels * sizeof(Vector3), hipMemcpyDeviceToHost));

		// Clean up device memory
		HANDLE_ERROR(hipFree(dev_Ls));
		HANDLE_ERROR(hipFree(dev_spheres));

		WritePPM(w, h, Ls);

		// Clean up host memory
		free(Ls);
	}
}

int main() {
    const uint32_t w = 1024u;
    const uint32_t h = 768u;
	int block_size = 8;
    FILE* log_fp = fopen("performance_log.csv", "w");
    std::fprintf(log_fp, "block_x,pixels_per_thread,samples,kernel_time_ms,total_time_ms,efficiency\n");
	for (int pixels_per_thread : {1}) {
		dim3 nthreads(block_size, block_size);
		int tile_size = block_size * sqrt(pixels_per_thread);  // 每个 block 实际处理的图像大小（像素数）
		dim3 nblocks((w + tile_size - 1) / tile_size,
					(h + tile_size - 1) / tile_size);

		for (int samples = 75; samples <= 100; samples += 200) {
			smallpt::Render(samples, nthreads, nblocks, log_fp, pixels_per_thread);
		}
	}

    fclose(log_fp);
    return 0;
}